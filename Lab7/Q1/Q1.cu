#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *t1, int *t2)
{
int n=threadIdx.x, m=blockIdx.x, size=blockDim.x;
a[m*size+n]=t1[m*size+n]+t2[m*size+n];
}
int main(void)
{
int *a,*t1, *t2, m,n,i,j;
int *d_a,*d_t1, *d_t2;
printf("Enter the value of m: ");scanf("%d",&m);
printf("Enter the value of n: ");scanf("%d",&n);
int size=sizeof(int)*m*n;
a=(int*)malloc(m*n*sizeof(int));
t1=(int*)malloc(m*n*sizeof(int));
t2=(int*)malloc(m*n*sizeof(int));
printf("Enter input matrix:\n");
for(i=0;i<m*n;i++)
scanf("%d",&t1[i]);
printf("Enter input matrix:\n");
for(i=0;i<m*n;i++)
scanf("%d",&t2[i]);
hipMalloc((void**)&d_a,size);
hipMalloc((void**)&d_t1,size);
hipMalloc((void**)&d_t2,size);
hipMemcpy(d_t1,t1,size,hipMemcpyHostToDevice);
hipMemcpy(d_t2,t2,size,hipMemcpyHostToDevice);
add<<<m,n>>>(d_a,d_t1, d_t2);
hipMemcpy(a,d_a,size,hipMemcpyDeviceToHost);
printf("Result vector is:\n");
for(i=0;i<n;i++)
{
for(j=0;j<m;j++)
printf("%d\t",a[i*m+j]);
printf("\n");
}
getchar();
hipFree(d_a);
hipFree(d_t1);
hipFree(d_t2);
}