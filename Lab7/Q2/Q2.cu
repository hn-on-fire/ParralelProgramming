#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 4
__global__ void matMultiply(float *A, float *B, float *C, int rowsA, int colsA, int rowsB, int colsB) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.x * blockDim.y + threadIdx.y;
    printf("i  = %d, j = %d\n", threadIdx.y, blockDim.y);
    if (i >= rowsA || j >= colsB) return;
    float sum = 0;
    for (int k = 0; k < rowsB; ++k) {
        sum += A[i * rowsB + k] * B[k * colsB + j];
    }
    C[i * colsB + j] = sum;
}

int main() {
    int rows = 4;
    int cols = 4;
    size_t size = sizeof(float)*rows*cols;
    float *A, *B, *C, *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            A[i * cols + j] = i % 2 == 0 ? 1.0f : 1.0f;
        }
    }
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            B[i * cols + j] = i % 2 == 0 ? 1.0f : 1.0f;
        }
    }

    hipMemcpy(d_A, A, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    int blocksPerGrid = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 gridSize(blocksPerGrid, 1, 1);
    dim3 blockSize(rows, cols, 1);

    matMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, rows, cols, rows, cols);

    hipMemcpy(C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", C[i * cols + j] );
        }
        printf("\n");
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}