#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 1

__global__ void countWords(int* wordFrequencies, char* sentence, int sentenceLength, char* word, int wordLength) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= sentenceLength-wordLength) return;

    char currChar = sentence[idx];
    int wordStart = 0;
    for(int i =0; i<wordLength; i++){
        if(word[i]!= sentence[idx+i])
        return;
    }
    atomicAdd(wordFrequencies, 1);
}

int main() {
    char sentence[] = "The quick brown fox jumps over the lazy dog", *d_s;
    int sentenceLength = strlen(sentence), d_sl;
    char word[] = "fox", *d_word;
    int wordLength = strlen(word), *d_wl;

    int wordFrequencies, *d_wf;
    hipMalloc((void**)&d_wf, sizeof(int));
    hipMalloc((void**)&d_word, sizeof(char)*wordLength);
    hipMalloc((void**)&d_s, sizeof(char)*sentenceLength);
    hipMalloc((void**)&d_sl, sizeof(int));
    int numBlocks = (sentenceLength + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipMemcpy(d_s, sentence, sizeof(char)*sentenceLength, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(char)*wordLength, hipMemcpyHostToDevice);
    hipMemcpy(d_wf, wordFrequencies, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(char)*wordLength, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(char)*wordLength, hipMemcpyHostToDevice);
    countWords<<<numBlocks, BLOCK_SIZE>>>(&wordFrequencies,sentence, sentenceLength, word, wordLength);

    hipDeviceSynchronize();

    

    printf("Max frequency of %s in %s is %d\n", word, sentence, maxFrequency);

    hipFree(wordFrequencies);
    return 0;
}