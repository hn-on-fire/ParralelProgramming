#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void mul(int* a1, int* a2, int* o, int len){
    int n =threadIdx.x;
    o[n] = a1[n]*a2[n];
}

int main(){
    printf("Enter mat dim: ");
    int m,n;
    scanf("%d %d", &m,&n);
    int l1 = 0, l2 = 0;
    int a1[20], a2[20];
    printf("Enter Elements");
    for (int i = 0; i < m*n; i++)
    {
        scanf("%d", &a1[l1]);
        if(a1[l1]!=0){
            l1++;
        }
    }
    printf("Enter mat dim: ");
    scanf("%d %d", &m,&n);
    printf("Enter Elements");
    for (int i = 0; i < m*n; i++)
    {
        scanf("%d", &a2[l2]);
        if(a2[l2]!=0){
            l2++;
        }
    }

    int *a1_d, *a2_d, *o;
    int o_h[20];
    hipMalloc((void**)&a1_d, sizeof(a1));
    hipMalloc((void**)&a2_d, sizeof(a1));
    hipMalloc((void**)&o, sizeof(a1));
    hipMemcpy(a1_d, a1, sizeof(a1), hipMemcpyHostToDevice);
    hipMemcpy(a2_d, a2, sizeof(a2), hipMemcpyHostToDevice);
    fflush(stdout);
    mul<<<1, l1>>>(a1_d,a2_d, o, l1);
    hipMemcpy(o_h, o, sizeof(a1), hipMemcpyDeviceToHost);
    for(int i =0;i<l1;i++){
        printf("%d ", o_h[i]);
    }
    printf("\n");
    fflush(stdout);
}