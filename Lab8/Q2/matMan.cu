#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void mul(int* a,int len){
    int n = threadIdx.y*len + threadIdx.x;
    if(threadIdx.y == 1)
        a[n] = a[n]*a[n];
    else if(threadIdx.y == 2)
        a[n] = a[n]*a[n]*a[n];
}

int main(){
    printf("Enter mat dim: ");
    int m,n;
    scanf("%d %d", &m,&n);
    int a[20];
    printf("Enter Elements");
    for (int i = 0; i < m*n; i++)
    {
        scanf("%d", &a[i]);
    }
    int *a_d;
    hipMalloc((void**)&a_d, sizeof(a));
    hipMemcpy(a_d, a, sizeof(a), hipMemcpyHostToDevice);
    dim3 mat(3,3);
    mul<<<1, mat>>>(a_d,n);
    hipMemcpy(a, a_d, sizeof(a), hipMemcpyDeviceToHost);
    for(int i =0;i<n*m;i++){
        printf("%d ", a[i]);
        if((i+1)%m ==0)
            printf("\n");
    }
    printf("\n");
    fflush(stdout);
}