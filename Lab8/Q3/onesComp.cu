#include <stdio.h>
#include "hip/hip_runtime.h"
#include <bitset>
__global__ void mul(int* a,dim3 mat ){
    int n = threadIdx.y * mat.x + threadIdx.x;
    if(threadIdx.y != 0 && threadIdx.x != 0 && threadIdx.y != mat.y -1 && threadIdx.x != mat.x -1)
        a[n] =  (int) std::bitset<8>(~a[n]).to_ulong();
}
int main(){
    printf("Enter mat dim: ");
    int m,n;
    scanf("%d %d", &m,&n);
    int a[20];
    printf("Enter Elements");
    for (int i = 0; i < m*n; i++)
    {
        scanf("%d", &a[i]);
    }
    int *a_d;
    hipMalloc((void**)&a_d, sizeof(a));
    hipMemcpy(a_d, a, sizeof(a), hipMemcpyHostToDevice);
    dim3 mat(m,n);
    mul<<<1, mat>>>(a_d,mat);
    hipMemcpy(a, a_d, sizeof(a), hipMemcpyDeviceToHost);
    for(int i =0;i<n*m;i++){
        printf("5d ", a[i]);
        if((i+1)%m ==0)
            printf("\n");
    }
    printf("\n");
    fflush(stdout);
}