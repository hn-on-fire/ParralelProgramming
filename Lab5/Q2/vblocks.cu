
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__
void add_vectors(double *a, double *b, double *c)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < N) c[id] = a[id] + b[id];
}
int main()
{
	size_t bytes = N*sizeof(double);
	double *A = (double*)malloc(bytes);
	double *B = (double*)malloc(bytes);
	double *C = (double*)malloc(bytes);

	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);
	for(int i=0; i<N; i++)
	{
		A[i] = 1.0;
		B[i] = 2.0;
	}
	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);
	add_vectors<<< ceil( float(N) / 256 ) ,256 >>>(d_A, d_B, d_C);

	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

	free(A);
	free(B);
	free(C);
	for(int i =0;i<N;i++){
		printf(" %lf %lf %lf ",A[i], B[i], C[i]);
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	printf("\n---------------------------\n");
	printf("__SUCCESS__\n");


	return 0;
}