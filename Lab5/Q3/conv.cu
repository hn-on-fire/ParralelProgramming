#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 4
#define INPUT_SIZE 12
#define MASK_WIDTH 5
__constant__ float M[MASK_WIDTH];

__global__ void conv(float *N, float *M, float *P, int Width){
	int i = blockIdx.x*blockDim.x+ threadIdx.x;
	float Pvalue = 0;
	int n_start_point = i-(MASK_WIDTH/2);
	for(int j =0; j<MASK_WIDTH;j++){
		if(n_start_point+j >=0 && n_start_point+j < Width){
			Pvalue+= N[n_start_point+j]*M[j];
		}
	}
	P[i]=Pvalue;
}

int main(){

	float *d_N = 0;
	float *d_P = 0;
    float *d_M = 0;
    float h_M[MASK_WIDTH]={2,3,4,3,2};
	hipMalloc(&d_N,INPUT_SIZE*sizeof(float));
	hipMalloc(&d_P,INPUT_SIZE*sizeof(float));
    hipMalloc(&d_M,MASK_WIDTH*sizeof(float));
	float *h_N = (float*)malloc(INPUT_SIZE*sizeof(float));
	float *h_P = (float*)malloc(INPUT_SIZE*sizeof(float));
	//float *h_M = (float*)malloc(MASK_WIDTH*sizeof(float));

	for(int i=0;i<INPUT_SIZE;++i){
		h_N[i]=(float)1;
	}
    
	hipMemcpy(d_N,h_N,INPUT_SIZE*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_P,h_P,INPUT_SIZE*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_M,h_M,MASK_WIDTH*sizeof(float),hipMemcpyHostToDevice);
	
    conv<<<(INPUT_SIZE+TILE_SIZE-1)/TILE_SIZE,TILE_SIZE >>>(d_N,d_M,d_P, MASK_WIDTH);
	hipMemcpy(h_P,d_P,INPUT_SIZE*sizeof(float),hipMemcpyDeviceToHost);
	for(int i=0; i<INPUT_SIZE;++i){
		printf("%f\n", h_P[i]);
	}
	hipFree(d_N);
	hipFree(d_P);
	hipFree(M);
	free(h_N);
	free(h_P);
	//free(h_M);
    return 0;
}